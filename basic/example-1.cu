#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void gpuFunc()
{
	printf("\n********************************************************\n");
	printf("*                 SOY LA GPU: \"Hello World\"          *\n");
	printf("********************************************************\n\n");

} // gpuFunc


/* Definicion de constantes */
#define currentGPU 0


int main()
{

	/* Indicamos la GPU (DEVICE) que vamos a utilizar */
	hipSetDevice(currentGPU);

	/* *******************************************
	 * Lanzamos la función del DEVICE (el Kernel)
	 *  1º parámetro: número de bloques (de hilos)
	 *  2º parámetro: número de hilos por bloque
	 * ***************************************** */
	gpuFunc<<<1, 1>>>();

	/* Liberar recursos del DEVICE */
	hipDeviceReset();

} // Fin main
