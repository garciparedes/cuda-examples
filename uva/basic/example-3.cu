/*
*
* Name: Exercise 3
* Subject: Parallel Computing (Degree on Computer Engineering)
* Institution: University of Valladolid
* @author: Gonzalez Escribano, Arturo
* @version: 1.0
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


/*Definición de constantes*/
#define currentGPU 0		//El número más alto suele indicar la salida de video



int main()
{

	/*Consultamos el número de GPUs*/
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	printf("El sistema tiene %d GPUs\n",deviceCount);

	/***********************************************************************
	*
	*   ANSWER_3_5:
	*
	*   GeForce GTX TITAN X: 	1024 1024 64 threads per block
	*   GeForce GTX TITAN Black:1024 1024 64 blocks per dim
	*
	***********************************************************************/


	/*Consultamos las características de las GPUs*/
	int device;
	for (device = 0; device < deviceCount; device++) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, device);
		printf("Device %d:\n",device);
		printf("\t\t\t Name: %s\n",deviceProp.name);
		printf("\t\t\t GPU architecture: %d.%d\n",deviceProp.major, deviceProp.minor);
		printf("\t\t\t multiProcessorCount: %d\n",deviceProp.multiProcessorCount);
		printf("\t\t\t totalGlobalMem: %.0f\n",(float) deviceProp.totalGlobalMem);
		printf("\t\t\t maxThreadsPerBlock: %d\n", deviceProp.maxThreadsPerBlock);
		printf("\t\t\t maxThreadsDim: %d %d %d\n", deviceProp.maxThreadsDim[0],
			deviceProp.maxThreadsDim[1],deviceProp.maxThreadsDim[2]);
		printf("\n\n");
	}
} //main
