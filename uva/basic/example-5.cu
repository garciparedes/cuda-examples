/*
*
* Name: Exercise 5
* Subject: Parallel Computing (Degree on Computer Engineering)
* Institution: University of Valladolid
* @author: Gonzalez Escribano, Arturo
* @author: García Prado, Sergio (@garciparedes)
* @version: 1.1
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>




/*Definición de constantes*/
#define currentGPU 0		//El número más alto suele indicar la salida de video
#define ELEMENTS 24



__global__ void  kernel_A (int *array){

    int global_id = threadIdx.x + threadIdx.y * blockDim.x +
        blockIdx.x * blockDim.x * blockDim.y +
        blockIdx.y * blockDim.x * blockDim.y * gridDim.x;

	array[ global_id ]= global_id +1 ;

}	//kernel_A

__global__ void  kernel_B (int *array){

    int global_id = threadIdx.x + threadIdx.y * blockDim.x +
        blockIdx.x * blockDim.x * blockDim.y +
        blockIdx.y * blockDim.x * blockDim.y * gridDim.x;

	array[ global_id ]= array[ global_id ] +1 ;

}	//kernel_B



int main()
{

	/*Indicamos la GPU (DEVICE) que vamos a utilizar*/
	hipSetDevice(currentGPU);

    /*Declaración del shape de los bloques y del Grid*/
    dim3 bloqShapeGpu(4,2);
    dim3 gridShapeGpu(3,1);

	/* Variables*/
	int *arrayHost;
	int *arrayDevice;

	/* Reservas de memoria HOST y DEVICE*/
	arrayHost = (int*) malloc(sizeof(int) * ELEMENTS);
	hipMalloc( (void**) &arrayDevice, sizeof(int) * (int) ELEMENTS);

	/* Inicialización e impresión inicial*/
	printf("\n\nArray inicialiado:");
	for(int i=0; i<ELEMENTS; i++){
		arrayHost[i]=1;
		printf(" %d ",arrayHost[i]);
	}
	printf("\n\n");

	/* Transferencia a memoria Device*/
	hipMemcpy(arrayDevice,arrayHost, sizeof(int) * ELEMENTS,hipMemcpyHostToDevice);

	/* Lanzamiento del primer kernel*/
	kernel_A<<<gridShapeGpu, bloqShapeGpu>>>(arrayDevice);

	hipDeviceSynchronize();

	/* Lanzamiento del segundo kernel*/
	kernel_B<<<gridShapeGpu, bloqShapeGpu>>>(arrayDevice);

	/* Transferencia a memoria Host*/
	hipMemcpy(arrayHost,arrayDevice, sizeof(int) * ELEMENTS,hipMemcpyDeviceToHost);


	/* Impresión */
	printf("Array despues del segundo kernel:");
	for(int i=0; i<ELEMENTS; i++){
		printf(" %d ",arrayHost[i]);
	}
	printf("\n\n");

	/*Liberamos memoria del DEVICE*/
	hipFree(arrayDevice);

	/*Liberamos los hilos del DEVICE*/
	hipDeviceReset();

} //main
