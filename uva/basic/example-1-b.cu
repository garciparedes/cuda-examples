/*
*
* Name: Exercise 1-b
* Subject: Parallel Computing (Degree on Computer Engineering)
* Institution: University of Valladolid
* @author: Gonzalez Escribano, Arturo
* @author: García Prado, Sergio (@garciparedes)
* @version: 1.1.1
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void gpuFunc()
{
	printf("\n********************************************************\n");
	printf("*                 SOY LA GPU: \"Hello World\"            *\n");
	printf("********************************************************\n\n");

} // gpuFunc


/* Definicion de constantes */
#define currentGPU 7


int main()
{
	hipError_t error;

	/* Indicamos la GPU (DEVICE) que vamos a utilizar */
	if( (error = hipSetDevice(currentGPU)) != hipSuccess){
	      printf("cuda-error: %s\n", hipGetErrorString( error ) );
	}

	/* *******************************************
	 * Lanzamos la función del DEVICE (el Kernel)
	 *  1º parámetro: número de bloques (de hilos)
	 *  2º parámetro: número de hilos por bloque
	 * ***************************************** */
	gpuFunc<<<1, 1>>>();

	/* Liberar recursos del DEVICE */
	hipDeviceReset();

} // Fin main
