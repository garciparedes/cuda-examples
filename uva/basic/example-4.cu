/*
*
* Name: Exercise 4
* Subject: Parallel Computing (Degree on Computer Engineering)
* Institution: University of Valladolid
* @author: Gonzalez Escribano, Arturo
* @author: García Prado, Sergio (@garciparedes)
* @version: 1.1
*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>




/*Definición de constantes*/
#define currentGPU 0		//El número más alto suele indicar la salida de video
#define ELEMENTS 5



__global__ void  kernel_A (int *array){
    array[ threadIdx.x ] = threadIdx.x;

    if (ELEMENTS % 2 == 1 && threadIdx.x == (ELEMENTS / 2)){
        array[ threadIdx.x ] = array[ threadIdx.x ];
    } else if (threadIdx.x < ((ELEMENTS / 2))){
        array[ threadIdx.x ] = array[ threadIdx.x + 1]*100 + array[ threadIdx.x ];
    } else{
        array[ threadIdx.x ] = array[ threadIdx.x - 1]*100 - array[ threadIdx.x ];
    }

}	//kernel_A




int main()
{

    hipError_t error;

	/*Indicamos la GPU (DEVICE) que vamos a utilizar*/
    if( (error = hipSetDevice(currentGPU)) != hipSuccess){
        printf("cuda-error-1: %s\n", hipGetErrorString( error ) );
        exit(-1);
    }
	/* Variables*/
	int *arrayHost;
	int *arrayDevice;

	/* Reservas de memoria HOST y DEVICE*/
	arrayHost = (int*) malloc(sizeof(int) * ELEMENTS);
    if( (error = hipMalloc( (void**) &arrayDevice, sizeof(int) * (int) ELEMENTS))
        != hipSuccess){

        printf("cuda-error-2: %s\n", hipGetErrorString( error ) );
        exit(-1);
    }
	/* Inicialización e impresión inicial*/
	printf("Array inicialiado:");
	for(int i=0; i<ELEMENTS; i++){
		arrayHost[i]=0;
		printf(" %d ",arrayHost[i]);
	}
	printf("\n");

	/* Transferencia a memoria Device*/
    if( (error = hipMemcpy(arrayDevice,arrayHost, sizeof(int)* ELEMENTS,hipMemcpyHostToDevice))
        != hipSuccess){

        printf("cuda-error-3: %s\n", hipGetErrorString( error ) );
        exit(-1);
    }

	/* Lanzamiento del kernel*/
	kernel_A<<<1,ELEMENTS>>>(arrayDevice);

    if( (error = hipGetLastError()) != hipSuccess){
        printf("cuda-error-4: %s\n", hipGetErrorString( error ) );
        exit(-1);
    }
	/* Transferencia a memoria Host*/
    if( (error = hipMemcpy(arrayHost,arrayDevice, sizeof(int)*ELEMENTS,hipMemcpyDeviceToHost))
        != hipSuccess){

        printf("cuda-error-5: %s\n", hipGetErrorString( error ) );
        exit(-1);
    }

	/* Inicialización e impresión inicial*/
	printf("Array despues del kernel:");
	for(int i=0; i<ELEMENTS; i++){
		printf(" %d ",arrayHost[i]);
	}
	printf("\n");

	/*Liberamos memoria del DEVICE*/

    if( (error = hipFree(arrayDevice)) != hipSuccess){
        printf("cuda-error-6: %s\n", hipGetErrorString( error ) );
        exit(-1);
    }
	/*Liberamos los hilos del DEVICE*/
    if( (error = hipDeviceReset()) != hipSuccess){
        printf("cuda-error-7: %s\n", hipGetErrorString( error ) );
        exit(-1);
    }

} //main
