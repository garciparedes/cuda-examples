/*
*
* Name: Exercise 2 - Main
* Subject: Parallel Computing (Degree on Computer Engineering)
* Institution: University of Valladolid
* @author: Gonzalez Escribano, Arturo
* @author: García Prado, Sergio (@garciparedes)
* @version: 1.1
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "kernel.cu"

/* Definición de constantes */
#define currentGPU 0		//El número más alto suele indicar la salida de video



int main()
{

	/*Indicamos la GPU (DEVICE) que vamos a utilizar*/
	hipSetDevice(currentGPU);

	/*Declaración del shape de los bloques y del Grid*/
		/*Primer Kernel*/
		dim3 bloqShapeGpuFunc1(3,2);	//bloques (de hilos): 3 columnas 2 filas;
		dim3 gridShapeGpuFunc1(2,3);	//grid (de bloques): 2 columnas 3 filas;

		/*Segundo Kernel*/
		dim3 bloqShapeGpuFunc2(256,2);	  //bloques (de hilos): 256 columnas 2 filas;
		dim3 gridShapeGpuFunc2(10,25);    //grid (de bloques): 10 columnas 25 filas;

	/*Funciones del DEVICE*/
		printf("Lanzamos el primer kernel...\n");

        /***********************************************************************
        *
        *   ANSWER_2_3:
        *
        *   (3*2) * (3*2) = 36 threads
        *
        ***********************************************************************/

        /*Primer Kernel*/
		gpuFunc1<<<gridShapeGpuFunc1, bloqShapeGpuFunc1>>>();

		printf("\n Lanzamos el segundo kernel...\n");

        /***********************************************************************
        *
        *   ANSWER_2_3:
        *
        *   (10*25) * (256*2) = 128000 threads
        *
        ***********************************************************************/

        /*Segundo Kernel*/
		gpuFunc2<<<gridShapeGpuFunc2, bloqShapeGpuFunc2>>>();

	/*Liberamos recursos del DEVICE*/
	hipDeviceReset();

} //Fin main
