
#include <hip/hip_runtime.h>
/*
*
* Name: Exercise 2 - Kernels
* Subject: Parallel Computing (Degree on Computer Engineering)
* Institution: University of Valladolid
* @author: Gonzalez Escribano, Arturo
* @author: García Prado, Sergio (@garciparedes)
* @version: 1.1
*/

/* ********************************************************************
 *
 * **************************   Kernels   ***************************
 *
 * ****************************************************************** */


/*Primer kernel*/
__global__ void gpuFunc1()
{
        /***********************************************************************
        *
        *   ANSWER_2_5:
        *
        *   (Code Below)
        *
        ***********************************************************************/
        int global_id = threadIdx.x + threadIdx.y * blockDim.x +
            blockIdx.x * blockDim.x * blockDim.y +
            blockIdx.y * blockDim.x * blockDim.y * gridDim.x;

        printf("\t Soy el hilo %d %d del bloque %d %d con id %d\n",
            threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y, global_id);

} // gpuFunc1


/************************************************************************/


/*Segundo kernel*/
__global__ void gpuFunc2()
{
        /***********************************************************************
        *
        *   ANSWER_2_4:
        *
        *   int global_id = threadIdx.x + threadIdx.y * blockDim.x +
        *       blockIdx.x * blockDim.x * blockDim.y +
        *       blockIdx.y * blockDim.x * blockDim.y * gridDim.x;
        *
        ***********************************************************************/

        /***********************************************************************
        *
        *   ANSWER_2_6:
        *
        *   (Code Below)
        *
        ***********************************************************************/

        int global_id = threadIdx.x + threadIdx.y * blockDim.x +
            blockIdx.x * blockDim.x * blockDim.y +
            blockIdx.y * blockDim.x * blockDim.y * gridDim.x;

        if (global_id == 55555){
        	printf("\t El shape del bloque de hilos es: %d columnas y  %d filas\n", blockDim.x,blockDim.y);
	        printf("\t El shape del Grid es: %d columnas  y %d filas\n", gridDim.x,gridDim.y);
	} // if

} // gpuFunc2
